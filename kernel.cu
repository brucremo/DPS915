#include "hip/hip_runtime.h"
//exercise 5.20 Deitel and deitel pag 242
//file: triangle_main.cpp
//file containing main function
#include "triangle.h" //include definition of class triangle
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <string>
#include <iostream>
#include <algorithm>
#include <cmath> //for the power function
#include <chrono>
#include <vector> 
#include <hip/hip_runtime.h>
#include <>
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/device_functions.h>
using namespace std;
using namespace std::chrono;

// constants in the program
const int EXPONENT = 2;
const int ntpb = 512;

__global__ void valueCheck(float value, float * arr, float * check, int size) {

	int idx = threadIdx.x;

	if (idx < size) {

		for (int i = 0; i < size; i++) {

			if ( (powf(value, EXPONENT) + powf(arr[i], EXPONENT) ) == pow(check[idx], EXPONENT)) {

				printf("Triple: ( %f  %f  %f )\n", value, arr[i], check[idx]);
			}
		}

		__syncthreads();
	}
}

/*the constructors initialize the values of the sides to 1 and pass them to the
calculateDimensions functions */
Triangle::Triangle(double Side1, double Side2, double Hypot, int hypotMaxSize){

	calculateDimensions(Side1, Side2, Hypot, hypotMaxSize);
}

float genToHundred() {

	static float i = 0;
	return ++i;
}

//set the 3 dimensions and calculates the Pythagorean triple
void Triangle::calculateDimensions(double Side1, double Side2, double Hypot, int hypotMaxSize){

	//Gathering device properties and calculating blocks and grids
	int d;
	hipDeviceProp_t prop;
	hipGetDevice(&d);
	hipGetDeviceProperties(&prop, d);
	unsigned ntpb = prop.maxThreadsDim[0];
	unsigned ntpg = ntpb * prop.maxGridSize[0];

	if (hypotMaxSize > ntpg) {
		hypotMaxSize = ntpg;
		std::cout << "hypotMaxSize reduced to " << hypotMaxSize << std::endl;
	}

	float * d_s2;
	hipMalloc((void**)&d_s2, hypotMaxSize * sizeof(float));

	float * d_Hyp;
	hipMalloc((void**)&d_Hyp, hypotMaxSize * sizeof(float));

	vector<float> v1(hypotMaxSize);
	generate(v1.begin()++, v1.end(), genToHundred);

	hipMemcpy(d_s2, v1.data(), hypotMaxSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Hyp, v1.data(), hypotMaxSize * sizeof(float), hipMemcpyHostToDevice);

	float * pos = v1.data();

	for (int i = 0; i < hypotMaxSize; i++) {

		valueCheck << <1, hypotMaxSize >> > (*pos++, d_s2, d_Hyp, hypotMaxSize);
	}
}

void Triangle::printDimensions(double side1, double side2, double hypotenuse){

	cout << "( " << side1 << " , " << side2 << " , " << hypotenuse << " ) " << endl;
}

int main(int argc, char * argv[]) {

	//Timing start
	auto start = high_resolution_clock::now();

	Triangle rightTriangle(std::stof(argv[1]), std::stof(argv[2]), std::stof(argv[3]), std::stoi(argv[4])); //create Triangle object
	hipDeviceSynchronize();

	//Timing capture
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);

	// To get the value of duration use the count() 
	// member function on the duration object 
	cout << duration.count() << endl;

	return 0;
}