#include "hip/hip_runtime.h"
//exercise 5.20 Deitel and deitel pag 242
//file: triangle_main.cpp
//file containing main function
#include "triangle.h" //include definition of class triangle
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <string>
#include <iostream>
#include <algorithm>
#include <cmath> //for the power function
#include <chrono>
#include <vector> 
#include <hip/hip_runtime.h>
#include <>
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/device_functions.h>
using namespace std;
using namespace std::chrono;

// constants in the program
const int EXPONENT = 2;
const int ntpb = 1024;

__global__ void valueCheck(float value, float * arr, float * check, int size) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) {

		for (int i = 0; i < size; i++) {

			if ( (pow(value, EXPONENT) + pow(arr[i], EXPONENT) ) == pow(check[idx], EXPONENT)) {

				printf("Triple: ( %f  %f  %f )\n", value, arr[i], check[idx]);
			}
		}

		__syncthreads();
	}
}

/*the constructors initialize the values of the sides to 1 and pass them to the
calculateDimensions functions */
Triangle::Triangle(double Side1, double Side2, double Hypot, int hypotMaxSize){

	calculateDimensions(Side1, Side2, Hypot, hypotMaxSize);
}

float gen() {

	static float i = 0;
	return ++i;
}

//set the 3 dimensions and calculates the Pythagorean triple
void Triangle::calculateDimensions(double Side1, double Side2, double Hypot, int hypotMaxSize){

	//Gathering device properties and calculating blocks and grids
	int d;
	hipDeviceProp_t prop;
	hipGetDevice(&d);
	hipGetDeviceProperties(&prop, d);
	unsigned ntpb = prop.maxThreadsDim[0];
	unsigned ntpg = ntpb * prop.maxGridSize[0];

	if (hypotMaxSize > ntpg) {
		hypotMaxSize = ntpg;
		std::cout << "hypotMaxSize reduced to " << hypotMaxSize << std::endl;
	}

	//Arrays allocated on the device
	float * d_Hyp;
	float * d_s2;
	hipMalloc((void**)&d_s2, hypotMaxSize * sizeof(float));
	hipMalloc((void**)&d_Hyp, hypotMaxSize * sizeof(float));

	//Initializing host vector from 1 to hypotMaxSize
	vector<float> h_v(hypotMaxSize);
	generate(h_v.begin()++, h_v.end(), gen);

	hipMemcpy(d_s2, h_v.data(), hypotMaxSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Hyp, h_v.data(), hypotMaxSize * sizeof(float), hipMemcpyHostToDevice);

	//Get initial data position on the vector
	float * pos = h_v.data();

	for (int i = 0; i < hypotMaxSize; i++) {

		valueCheck <<<(hypotMaxSize + ntpb - 1) / ntpb, ntpb >>> (*pos++, d_s2, d_Hyp, hypotMaxSize);
	}
}

void Triangle::printDimensions(double side1, double side2, double hypotenuse){

	cout << "( " << side1 << " , " << side2 << " , " << hypotenuse << " ) " << endl;
}

int main(int argc, char * argv[]) {

	//Timing start
	auto start = high_resolution_clock::now();

	Triangle rightTriangle(std::stof(argv[1]), std::stof(argv[2]), std::stof(argv[3]), std::stoi(argv[4])); //create Triangle object
	hipDeviceSynchronize();

	//Timing capture
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);

	// To get the value of duration use the count() 
	// member function on the duration object 
	cout << duration.count() << endl;

	return 0;
}