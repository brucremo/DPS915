#include "hip/hip_runtime.h"
//exercise 5.20 Deitel and deitel pag 242
//file: triangle_main.cpp
//file containing main function
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <string>
#include <iostream>
#include <algorithm>
#include <cmath> //for the power function
#include <chrono>
#include <time.h>
#include <vector> 
#include <hip/hip_runtime.h>
#include <>
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/device_functions.h>
using namespace std;
using namespace std::chrono;

// constants in the program
const int EXPONENT = 2;

__global__ void valueCheck(float value, float * arr, float * check, int size) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) {

		for (int i = 0; i < size; i++) {

			if ((pow(value, EXPONENT) + pow(arr[i], EXPONENT)) == pow(check[idx], EXPONENT)) {

				printf("Triple: ( %f  %f  %f )\n", value, arr[i], check[idx]);
			}
		}

		__syncthreads();
	}
}

void printDimensions(float side1, float side2, float hypotenuse) {

	cout << "Triple: ( " << side1 << " , " << side2 << " , " << hypotenuse << " ) " << endl;
}

float gen() {

	static float i = 0;
	return ++i;
}

//set the 3 dimensions and calculates the Pythagorean triple
void calculateCUDA(float Side1, float Side2, float Hypot, int hypotMaxSize) {

	//Gathering device properties and calculating blocks and grids
	int d;
	hipDeviceProp_t prop;
	hipGetDevice(&d);
	hipGetDeviceProperties(&prop, d);
	unsigned ntpb = prop.maxThreadsDim[0];
	unsigned ntpg = ntpb * prop.maxGridSize[0];

	if (hypotMaxSize > ntpg) {
		hypotMaxSize = ntpg;
		std::cout << "hypotMaxSize reduced to " << hypotMaxSize << std::endl;
	}

	//Arrays allocated on the device
	float * d_Hyp;
	float * d_s2;
	hipMalloc((void**)&d_s2, hypotMaxSize * sizeof(float));
	hipMalloc((void**)&d_Hyp, hypotMaxSize * sizeof(float));

	//Initializing host vector from 1 to hypotMaxSize
	vector<float> h_v(hypotMaxSize);
	generate(h_v.begin()++, h_v.end(), gen);

	hipMemcpy(d_s2, h_v.data(), hypotMaxSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Hyp, h_v.data(), hypotMaxSize * sizeof(float), hipMemcpyHostToDevice);

	//Get initial data position on the vector
	float * pos = h_v.data();

	// launch
	int nb = (hypotMaxSize + ntpb - 1) / ntpb;


	for (int i = 0; i < hypotMaxSize; i++) {

		valueCheck << <nb, ntpb >> > (*pos++, d_s2, d_Hyp, hypotMaxSize);
		hipDeviceSynchronize();
	}
}

void calculateSerial(float Side1, float Side2, float Hypot, int hypotMaxSize){

	//int i = 0 ; //for the iteration in the for loop
	for (int i = 1; i <= hypotMaxSize; i++){

		Side1 = i;
		for (int j = 1; j <= hypotMaxSize; j++){

			Side2 = j;
			for (int k = 1; k <= hypotMaxSize; k++){

				Hypot = k;
				if ((pow(Side1, EXPONENT)) + (pow(Side2, EXPONENT)) == (pow(Hypot, EXPONENT))){

					printDimensions(Side1, Side2, Hypot);
				}
			}
		}
	}
}

int main(int argc, char * argv[]) {

	//Timing start
	clock_t t;
	t = clock();

	calculateCUDA(std::stof(argv[1]), std::stof(argv[2]), std::stof(argv[3]), std::stoi(argv[4])); //create Triangle object

	//Timing capture
	t = clock() - t;

	//Timing start
	clock_t b;
	b = clock();

	calculateSerial(std::stof(argv[1]), std::stof(argv[2]), std::stof(argv[3]), std::stoi(argv[4])); //create Triangle object

	cout << "time on GPU: " << t * 1.0 / CLOCKS_PER_SEC << " seconds" << endl;

	//Timing capture
	b = clock() - b;
	cout << "time on CPU: " << b * 1.0 / CLOCKS_PER_SEC << " seconds" << endl;

	return 0;
}