#include "hip/hip_runtime.h"
//exercise 5.20 Deitel and deitel pag 242
//file: triangle_main.cpp
//file containing main function
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <string>
#include <iostream>
#include <algorithm>
#include <cmath> //for the power function
#include <chrono>
#include <time.h>
#include <vector> 
#include <hip/hip_runtime.h>
#include <>
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/device_functions.h>
using namespace std;
using namespace std::chrono;

// constants in the program
const int EXPONENT = 2;
//const int ntpb = 1024;

__global__ void valueCheck(float value, float * arr, float * check, const int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int t = threadIdx.x;
	__shared__ float s_check[ntpb];
	__shared__ float s_arr[ntpb];
	
	s_check[t] = check[idx];
	s_arr[t] = arr[idx];
	__syncthreads();

	if (idx < size) {

		for (int i = 0; i < size; i++) {

			if ((value * value) + (s_arr[i] * s_arr[i]) == (s_check[t] * s_check[t]) {

				printf("Triple: ( %f  %f  %f )\n", value, s_arr[i], s_check[t]);
			}
		}

		__syncthreads();
	}
}

void printDimensions(float side1, float side2, float hypotenuse) {

	cout << "Triple: ( " << side1 << " , " << side2 << " , " << hypotenuse << " ) " << endl;
}

float gen() {

	static float i = 0;
	return ++i;
}

//set the 3 dimensions and calculates the Pythagorean triple
void calculateCUDA(float Side1, float Side2, float Hypot, int hypotMaxSize) {

	//Gathering device properties and calculating blocks and grids
	int d;
	hipDeviceProp_t prop;
	hipGetDevice(&d);
	hipGetDeviceProperties(&prop, d);
	unsigned ntpb = prop.maxThreadsDim[0];
	unsigned ntpg = ntpb * prop.maxGridSize[0];

	if (hypotMaxSize > ntpg) {
		hypotMaxSize = ntpg;
		std::cout << "hypotMaxSize reduced to " << hypotMaxSize << std::endl;
	}

	//Arrays allocated on the device
	float * d_Hyp;
	float * d_s2;
	hipMalloc((void**)&d_s2, hypotMaxSize * sizeof(float));
	hipMalloc((void**)&d_Hyp, hypotMaxSize * sizeof(float));

	//Initializing host vector from 1 to hypotMaxSize
	vector<float> h_v(hypotMaxSize);
	generate(h_v.begin()++, h_v.end(), gen);

	hipMemcpy(d_s2, h_v.data(), hypotMaxSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Hyp, h_v.data(), hypotMaxSize * sizeof(float), hipMemcpyHostToDevice);

	//Get initial data position on the vector
	float * pos = h_v.data();

	// launch
	int nb = (hypotMaxSize + ntpb - 1) / ntpb;


	for (int i = 0; i < hypotMaxSize; i++) {

		valueCheck << <nb, ntpb >> > (*pos++, d_s2, d_Hyp, hypotMaxSize);
		hipDeviceSynchronize();
	}
}

void calculateSerial(float Side1, float Side2, float Hypot, int hypotMaxSize) {

	//int i = 0 ; //for the iteration in the for loop
	for (int i = 1; i <= hypotMaxSize; i++) {

		Side1 = i;
		for (int j = 1; j <= hypotMaxSize; j++) {

			Side2 = j;
			for (int k = 1; k <= hypotMaxSize; k++) {

				Hypot = k;
				if ((Side1 * Side1) + (Side2 * Side2) == (Hypot * Hypot)) {

					printDimensions(Side1, Side2, Hypot);
				}
			}
		}
	}
}

int main(int argc, char * argv[]) {

	//Timing start
	clock_t t;
	t = clock();

	calculateCUDA(std::stof(argv[1]), std::stof(argv[2]), std::stof(argv[3]), std::stoi(argv[4])); //create Triangle object

	//Timing capture
	t = clock() - t;

	//Timing start
	clock_t b;
	b = clock();

	calculateSerial(std::stof(argv[1]), std::stof(argv[2]), std::stof(argv[3]), std::stoi(argv[4])); //create Triangle object

	cout << "time on GPU: " << t * 1.0 / CLOCKS_PER_SEC << " seconds" << endl;

	//Timing capture
	b = clock() - b;
	cout << "time on CPU: " << b * 1.0 / CLOCKS_PER_SEC << " seconds" << endl;

	return 0;
}
